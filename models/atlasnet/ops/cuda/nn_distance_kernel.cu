#include "hip/hip_runtime.h"
/* CUDA Implementation for feature interpolation
Pytorch has different APIs for different versions.
v1.1.0: type() -> scalar_type() for dispatch
v1.2.0: AT_CHECK -> TORCH_CHECK
*/
#ifndef _NN_DISTANCE_KERNEL
#define _NN_DISTANCE_KERNEL

#include <cmath>
#include <vector>

#include <ATen/ATen.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>  // at::cuda::getApplyGrid
#include <THC/THC.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.type().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)


/********************************
* Forward kernel
*********************************/
template<typename scalar_t, typename index_t, uint64_t BLOCK_SIZE, uint64_t DIM>
__global__ void NNDistanceForwardKernel(
    const scalar_t *__restrict__ xyz1,
    const scalar_t *__restrict__ xyz2,
    scalar_t *__restrict__ dist,
    index_t *__restrict__ idx,
    const int64_t batch_size,
    const int64_t n1,
    const int64_t n2){
  // calculate the number of blocks
  const int64_t num_block1 = (n1 + BLOCK_SIZE - 1) / BLOCK_SIZE;
  const int64_t num_block2 = (n2 + BLOCK_SIZE - 1) / BLOCK_SIZE;
  const int64_t total_blocks = batch_size * num_block1;

  for (int block_idx = blockIdx.x; block_idx < total_blocks; block_idx += gridDim.x) {
    __shared__ scalar_t xyz2_buffer[BLOCK_SIZE * DIM];
    const int batch_idx = block_idx / num_block1;
    const int block_idx1 = block_idx % num_block1;
    const int xyz1_idx = (block_idx1 * BLOCK_SIZE) + threadIdx.x;
    const int xyz1_offset = (batch_idx * n1 + xyz1_idx) * DIM;
    scalar_t coords1[DIM] = {0.0};
    if (xyz1_idx < n1) {
      #pragma unroll
      for (int i = 0; i < DIM; ++i) {
        coords1[i] = xyz1[xyz1_offset + i];
      }
    } else {
      // x1 = y1 = z1 = 0.0;
      ;
    }
    scalar_t min_dist = 1e32;
    index_t min_idx = -1;
    // load a block of xyz2 data to reduce the times to read data
    for (int block_idx2 = 0; block_idx2 < num_block2; ++block_idx2) {
      // load xyz2 data
      int xyz2_idx = (block_idx2 * BLOCK_SIZE) + threadIdx.x;
      int xyz2_offset = (batch_idx * n2 + xyz2_idx) * DIM;
      if (xyz2_idx < n2) {
        #pragma unroll
        for (int i = 0; i < DIM; ++i) {
          xyz2_buffer[threadIdx.x * DIM + i] = xyz2[xyz2_offset + i];
        }
      }
      __syncthreads();
      // calculate the distance between xyz1 and xyz2, with the shared memory.
      for (int j = 0; j < BLOCK_SIZE; ++j) {
        xyz2_idx = (block_idx2 * BLOCK_SIZE) + j;
        const int buffer_offset = j * DIM;
        scalar_t d = 0.0;
        #pragma unroll
        for (int i = 0; i < DIM; ++i) {
          scalar_t coord = xyz2_buffer[buffer_offset + i];
          d += (coord - coords1[i]) * (coord - coords1[i]);
        }
        if (xyz2_idx < n2 && d < min_dist) {
          min_dist = d;
          min_idx = xyz2_idx;
        }
      }
      __syncthreads();
    }
    if (xyz1_idx < n1) {
      const int output_offset = batch_idx * n1 + xyz1_idx;
      dist[output_offset] = min_dist;
      idx[output_offset] = min_idx;
    }
  }
}

inline bool getGrid(uint64_t numBlocks, dim3& grid, int64_t curDevice) {
  if (curDevice == -1) return false;
  uint64_t maxGridX = at::cuda::getDeviceProperties(curDevice)->maxGridSize[0];
  if (numBlocks > maxGridX)
      numBlocks = maxGridX;
  grid = dim3(numBlocks);
  return true;
}

#define CASE_RUN(DIM) \
  case DIM: \
    AT_DISPATCH_FLOATING_TYPES(xyz1.scalar_type(), "NNDistanceForward", ([&] { \
      NNDistanceForwardKernel<scalar_t, int64_t, BLOCK_SIZE, DIM> \
        <<<grid1, BLOCK_SIZE>>>( \
          xyz1.data<scalar_t>(), \
          xyz2.data<scalar_t>(), \
          dist1.data<scalar_t>(), \
          idx1.data<int64_t>(), \
          batch_size, n1, n2); \
      })); \
    break;

/* Forward interface
Input:
  xyz1: (B, N1, 3)
  xyz2: (B, N2, 3)
Output:
  dist1: (B, N1)
  idx1: (B, N1)
  dist2: (B, N2)
  idx2: (B, N2)
*/
std::vector<at::Tensor> NNDistanceForward(
    const at::Tensor xyz1,
    const at::Tensor xyz2){
  const auto batch_size = xyz1.size(0);
  const auto n1 = xyz1.size(1);
  const auto n2 = xyz2.size(1);
  const auto dim = xyz1.size(2);

  CHECK_EQ(xyz2.size(0), batch_size);
  CHECK_EQ(xyz2.size(2), dim);
  CHECK_INPUT(xyz1);
  CHECK_INPUT(xyz2);

  auto dist1 = at::zeros({batch_size, n1}, xyz1.type());
  auto idx1 = at::zeros({batch_size, n1}, xyz1.type().toScalarType(at::kLong));

  // Calculate grids and blocks for kernels
  const uint64_t BLOCK_SIZE = 512;
  const auto num_block1 = (n1 + BLOCK_SIZE - 1) / BLOCK_SIZE;
  // From getApplyGrid: aten/src/ATen/cuda/CUDAApplyUtils.cuh
  dim3 grid1;
  const auto curDevice = at::cuda::current_device();
  getGrid(batch_size * num_block1, grid1, curDevice);

  // printf("(b, nb, n1, n2): (%ld, %ld, %ld, %ld)\n", batch_size, num_block1, n1, n2);
  
  switch (dim) {
    CASE_RUN(2)
    CASE_RUN(3)
    default:
      TORCH_CHECK(false, "Only support dim=2 or 3.");
  }
  THCudaCheck(hipGetLastError());

  return std::vector<at::Tensor>({dist1, idx1});
}


/**********************************
* Backward kernel
***********************************/
/* Backward Kernel */
template <typename scalar_t, typename index_t, uint64_t DIM>
__global__ void NNDistanceBackwardKernel(
    const scalar_t *__restrict__ grad_dist,
    const index_t *__restrict__ index,
    const scalar_t *__restrict__ xyz1,
    const scalar_t *__restrict__ xyz2,
    scalar_t *__restrict__ grad_xyz1,
    scalar_t *__restrict__ grad_xyz2,
    const int64_t batch_size,
    const int64_t n1,
    const int64_t n2) {
  const uint64_t totalElements = batch_size * n1;
  for (int linearId = blockIdx.x * blockDim.x + threadIdx.x;
       linearId < totalElements;
       linearId += gridDim.x * blockDim.x) {
    int batch_idx  = linearId / n1;
    int xyz1_offset = linearId * DIM;
    int xyz2_offset = (batch_idx * n2 + index[linearId])* DIM;
    scalar_t g = grad_dist[linearId] * 2;
    scalar_t coord1, coord2, g_tmp;
    #pragma unroll
    for (int i = 0; i < DIM; ++i) {
      coord1 = xyz1[xyz1_offset + i];
      coord2 = xyz2[xyz2_offset + i];
      g_tmp = g * (coord1 - coord2);
      atomicAdd(grad_xyz1 + xyz1_offset + i, g_tmp);
      atomicAdd(grad_xyz2 + xyz2_offset + i, -g_tmp);
    }
  }
}

#define CASE_RUN2(DIM) \
  case DIM: \
    AT_DISPATCH_FLOATING_TYPES(grad_dist1.scalar_type(), "NNDistanceBackward", ([&] { \
      NNDistanceBackwardKernel<scalar_t, int64_t, DIM> \
        <<<grid1, block>>>( \
          grad_dist1.data<scalar_t>(), \
          idx1.data<int64_t>(), \
          xyz1.data<scalar_t>(), \
          xyz2.data<scalar_t>(), \
          grad_xyz1.data<scalar_t>(), \
          grad_xyz2.data<scalar_t>(), \
          batch_size, n1, n2); \
    })); \
    break;

/* Backward interface
Input:
  grad_dist1: (B, N1)
  xyz1: (B, N1, 3)
  xyz2: (B, N2, 3)
  idx1: (B, N1)
Output:
  grad_xyz1: (B, N1, 3)
  grad_xyz2: (B, N2, 3)
*/
std::vector<at::Tensor> NNDistanceBackward(
    const at::Tensor grad_dist1,
    const at::Tensor xyz1,
    const at::Tensor xyz2,
    const at::Tensor idx1){
  const auto batch_size = grad_dist1.size(0);
  const auto n1 = xyz1.size(1);
  const auto n2 = xyz2.size(1);
  const auto dim = xyz1.size(2);
  CHECK_EQ(xyz1.size(0), batch_size);
  CHECK_EQ(xyz2.size(0), batch_size);
  CHECK_EQ(grad_dist1.size(1), n1);
  CHECK_EQ(xyz2.size(2), dim);
  CHECK_EQ(idx1.size(0), batch_size);
  CHECK_EQ(idx1.size(1), n1);
  CHECK_INPUT(grad_dist1);
  CHECK_INPUT(xyz1);
  CHECK_INPUT(xyz2);
  CHECK_INPUT(idx1);

  auto grad_xyz1 = at::zeros({batch_size, n1, dim}, xyz1.type());
  auto grad_xyz2 = at::zeros({batch_size, n2, dim}, xyz2.type());
  // Calculate grids and blocks for kernels
  const dim3 block = at::cuda::getApplyBlock();
  dim3 grid1;
  const auto curDevice = at::cuda::current_device();
  // getApplyGrid: aten/src/ATen/cuda/CUDAApplyUtils.cuh
  THArgCheck(at::cuda::getApplyGrid(batch_size * n1, grid1, curDevice), 1, "Too many elements to calculate");

  switch (dim) {
    CASE_RUN2(2)
    CASE_RUN2(3)
    default:
      TORCH_CHECK(false, "Only support dim=2 or 3.");
  }
  THCudaCheck(hipGetLastError());

  return std::vector<at::Tensor>({grad_xyz1, grad_xyz2});
}

#endif